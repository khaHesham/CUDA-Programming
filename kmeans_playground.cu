#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>

#include <hip/hip_runtime.h>


#define BLOCK_SIZE 128
#define CONVERGENCE_THRESHOLD 0
#define MAX_ITER 100

typedef unsigned int uint;

typedef struct Params
{
    uint N;
    uint D;
    uint K;
} Params;


// Kernel to compute distances between a datapoint and all centroids
__device__ float computeDistance(const float *datapoint, const float *centroid, int D)
{
    float distance = 0.0f;
    int tid = threadIdx.x;
    int blockSize = blockDim.x;

    // Compute distance in parallel
    for (int j = tid; j < D; j += blockSize)
    {
        float diff = datapoint[j] - centroid[j];
        distance += diff * diff;
    }

    // Reduce within the block using shared memory
    __shared__ float shared_distance[BLOCK_SIZE];
    shared_distance[tid] = distance;
    __syncthreads();

    for (int stride = blockSize / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            shared_distance[tid] += shared_distance[tid + stride];
        }
        __syncthreads();
    }

    // Store the final result in distance
    if (tid == 0)
    {
        distance = shared_distance[0];
    }

    return distance;
}

// Child kernel to find the nearest centroid for each datapoint
__global__ void findNearestCentroids(const float *datapoints, const float *centroids, uint *assignments, int N, int K, int D)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        float minDist = FLT_MAX;
        uint nearestCentroid = 0;
        for (int centroidId = 0; centroidId < K; ++centroidId)
        {
            float dist = computeDistance(&datapoints[idx * D], &centroids[centroidId * D], D);
            if (dist < minDist)
            {
                minDist = dist;
                nearestCentroid = centroidId;
            }
        }
        if (nearestCentroid != assignments[idx])
        {
            assignments[idx] = nearestCentroid;
        }
    }
}

__global__ void assign_points(float *datapoints, float *centroids, uint *assignments, Params params) {
     extern __shared__ char shared_mem[];

    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Since only one shared memory array is allowed, we divide it using pointer arithmetic
    float *centroids_s = (float *)shared_mem;
    float *datapoint_s = (float *)&centroids_s[params.K * params.D];

    // Cache the centroids in shared memory
    for (int cluster = threadIdx.x; cluster < params.K; cluster += blockDim.x)
    {
        for (int j = 0; j < params.D; j++)
            centroids_s[cluster * params.D + j] = centroids[cluster * params.D + j];
    }

    // Cache the datapoints since each will be read K times by a thread
    // Although no data is shared between threads, we still want to get the benefit of caching
    if (idx < params.N)
    {
        for (int j = 0; j < params.D; j++)
            datapoint_s[threadIdx.x * params.D + j] = datapoints[idx * params.D + j];
    }

    __syncthreads();

    // check if we have a multiple dimensions then run computeDistance kernell else run the simple distance calculation
    if (params.D > 10)
    {
        float min_dist = FLT_MAX;
        uint min_cluster = 0;

        for (int cluster = 0; cluster < params.K; cluster++)
        {
            float dist = computeDistance(&datapoint_s[threadIdx.x * params.D], &centroids_s[cluster * params.D], params.D);
            if (dist < min_dist)
            {
                min_dist = dist;
                min_cluster = cluster;
            }
        }

        if (min_cluster != assignments[idx])
            assignments[idx] = min_cluster;
    }
    else
    {
        float min_dist = FLT_MAX;
        uint min_cluster = 0;

        for (int cluster = 0; cluster < params.K; cluster++)
        {
            float dist = (datapoint_s[threadIdx.x] - centroids_s[cluster]) * (datapoint_s[threadIdx.x] - centroids_s[cluster]);
            if (dist < min_dist)
            {
                min_dist = dist;
                min_cluster = cluster;
            }
        }

        if (min_cluster != assignments[idx])
            assignments[idx] = min_cluster;
    }

}

__global__ void update_centroids(float *datapoints, uint *assignments, float *centroids, uint *clusters_count, Params params)
{
    // Define private centroids in shared memory (privatization + shared memory optimization)
    extern __shared__ char shared_mem[];

    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Since only one shared memory array is allowed, we divide it using pointer arithmetic
    float *centroids_s = (float *)shared_mem;
    uint *clusters_count_s = (uint *)&centroids_s[params.K * params.D];

    // Initialize both centroids and centroid counts to zeros
    for (int cluster = threadIdx.x; cluster < params.K; cluster += blockDim.x)
    {
        clusters_count_s[cluster] = 0;

        for (int j = 0; j < params.D; j++)
            centroids_s[cluster * params.D + j] = 0.0f;
    }

    __syncthreads();

    if (idx < params.N)
    {
        // Accumulate in the private shared arrays atomically
        uint cluster_id = assignments[idx];

        atomicAdd(&clusters_count_s[cluster_id], (uint)1);

        for (int j = 0; j < params.D; j++)
            atomicAdd(&centroids_s[cluster_id * params.D + j], datapoints[idx * params.D + j]);
    }

    __syncthreads();

    // Commit results in the global array atomically
    for (int cluster = threadIdx.x; cluster < params.K; cluster += blockDim.x)
    {
        atomicAdd(&clusters_count[cluster], clusters_count_s[cluster]);

        for (int j = 0; j < params.D; j++)
            atomicAdd(&centroids[cluster * params.D + j], centroids_s[cluster * params.D + j]);
    }
}

// Should use templates instead (later)
__global__ void set_to_zero(float *data, uint n)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        data[idx] = 0.0f;
}
__global__ void set_to_zero(uint *data, uint n)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        data[idx] = 0;
}

__global__ void divide(float *dividend, uint *divisor, uint n, uint D)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
    {
        uint div = divisor[idx];
        if (div != 0)
        {
            for (int j = 0; j < D; j++)
                dividend[idx * D + j] = dividend[idx * D + j] / div;
        }
    }
}

void initialize_centroids(float *centroids, float *datapoints, Params params)
{
    for (int i = 0; i < params.K; i++)
    {
        uint point_idx = rand() % params.N; // generate a random number between [0, N)
        for (int j = 0; j < params.D; j++)
            centroids[i * params.D + j] = datapoints[point_idx * params.D + j];
    }
    // for (int i = 0; i < params.K * params.D; i++)
    //     centroids[i] = (float)rand() / RAND_MAX;

    // FILE* init_centroids_file = fopen("init.txt", "r");
    // if (init_centroids_file == NULL) {
    //     printf("Error opening inital centroids file.\n");
    //     exit(1);
    // }

    // for (int i = 0; i < params.K * params.D; i++)
    //     fscanf(init_centroids_file, "%f", &centroids[i]);

    // fclose(init_centroids_file);

    // write initial centroids to a file
    FILE *init_centroids_file = fopen("init.txt", "w");
    if (init_centroids_file == NULL)
    {
        printf("Error opening inital centroids file.\n");
        exit(1);
    }

    for (int i = 0; i < params.K; i++)
    {
        for (int j = 0; j < params.D; j++)
            fprintf(init_centroids_file, "%f ", centroids[i * params.D + j]);
        fprintf(init_centroids_file, "\n");
    }
    fclose(init_centroids_file);
}

void write_results(float *centroids, uint *assignments, const char *clusters_path, const char *centroids_path, Params params)
{
    FILE *clusters_file = fopen(clusters_path, "w");
    if (clusters_file == NULL)
    {
        printf("Error opening clusters file.\n");
        exit(1);
    }

    for (int i = 0; i < params.N; i++)
        fprintf(clusters_file, "%d\n", assignments[i]);

    fclose(clusters_file);

    FILE *centroids_file = fopen(centroids_path, "w");
    if (centroids_file == NULL)
    {
        printf("Error opening centroids file.\n");
        exit(1);
    }

    for (int i = 0; i < params.K; i++)
    {
        for (int j = 0; j < params.D; j++)
            fprintf(centroids_file, "%f ", centroids[i * params.D + j]);
        fprintf(centroids_file, "\n");
    }

    fclose(centroids_file);
}

void kmeans(float *datapoints, float *centroids, uint *assignments, Params params)
{
    uint N = params.N, D = params.D, K = params.K;

    // Allocating device memory
    float *d_datapoints, *d_centroids;
    uint *d_assignments, *d_clusters_count;

    hipMalloc((void **)&d_datapoints, N * D * sizeof(float));
    hipMalloc((void **)&d_centroids, K * D * sizeof(float));
    hipMalloc((void **)&d_assignments, N * sizeof(uint));
    hipMalloc((void **)&d_clusters_count, K * sizeof(int));

    initialize_centroids(centroids, datapoints, params);

    // Copy data to device
    hipMemcpy(d_datapoints, datapoints, N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, K * D * sizeof(float), hipMemcpyHostToDevice);

    // Grid configuration (Should be reconsidered later)
    dim3 gridDim((N - 1) / BLOCK_SIZE + 1, 1);
    dim3 blockDim(BLOCK_SIZE, 1);

    size_t update_shared_mem = K * D * sizeof(float) + K * sizeof(uint);
    size_t assign_shared_mem = K * D * sizeof(float) + blockDim.x * D * sizeof(float);

    assign_points<<<gridDim, blockDim, assign_shared_mem>>>(d_datapoints, d_centroids, d_assignments, params);
    hipDeviceSynchronize();

    int iter = 0;
    while (iter < MAX_ITER)
    {
        set_to_zero<<<(K * D - 1) / BLOCK_SIZE + 1, blockDim>>>(d_centroids, K * D);
        set_to_zero<<<(K - 1) / BLOCK_SIZE + 1, blockDim>>>(d_clusters_count, K);
        update_centroids<<<gridDim, blockDim, update_shared_mem>>>(d_datapoints, d_assignments, d_centroids, d_clusters_count, params);
        divide<<<(K - 1) / BLOCK_SIZE + 1, blockDim>>>(d_centroids, d_clusters_count, K, D);

        assign_points<<<gridDim, blockDim, assign_shared_mem>>>(d_datapoints, d_centroids, d_assignments, params);

        hipError_t cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            printf("Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(1);
        }

        iter++;
    }

    hipMemcpy(centroids, d_centroids, K * D * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(assignments, d_assignments, N * sizeof(uint), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_datapoints);
    hipFree(d_centroids);
    hipFree(d_assignments);
    hipFree(d_clusters_count);
}

int main(int argc, char *argv[])
{
    if (argc < 7)
    {
        printf("Usage: ./kmeans data.txt clusters.txt centroids.txt N D K");
        return 1;
    }

    // set the seed for random number generation
    srand(0);

    uint N = atoi(argv[4]);
    uint D = atoi(argv[5]);
    uint K = atoi(argv[6]);
    Params params = {N, D, K};

    // Allocating host memory
    float *datapoints, *centroids;
    uint *assignments;

    datapoints = (float *)malloc(N * D * sizeof(float));
    centroids = (float *)malloc(K * D * sizeof(float));
    assignments = (uint *)malloc(N * sizeof(uint));

    // Read data points input file
    FILE *data_file = fopen(argv[1], "r");
    if (data_file == NULL)
    {
        printf("Error opening data file.\n");
        return 1;
    }

    for (int i = 0; i < N * D; i++)
        fscanf(data_file, "%f", &datapoints[i]);

    fclose(data_file);

    kmeans(datapoints, centroids, assignments, params);

    write_results(centroids, assignments, argv[2], argv[3], params);

    // Free host memory
    free(datapoints);
    free(centroids);
    free(assignments);

    return 0;
}
